
#include <hip/hip_runtime.h>
#include <iostream>

#define INDEX_NUM   3

#define INDEX_SUM   0
#define INDEX_MAX   1
#define INDEX_MIN   2

#define NUM_MAX         1024

#define ITEMS_NUM       (1024 * 1024)
#define BLOCK_SIZE      256

using namespace std;

__global__ void kernel_no_atomics(int *data, int *results) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        for (int i = 0; i < ITEMS_NUM; i++) {
            results[INDEX_SUM] += data[i];

            results[INDEX_MAX] = max(results[INDEX_MAX], data[i]);

            results[INDEX_MIN] = min(results[INDEX_MIN], data[i]);
        }
    }
}

__global__ void kernel_partial_atomics(int *data, int *results) {
    if (threadIdx.x == 0) {
        int start = blockIdx.x * BLOCK_SIZE;
        int aux[INDEX_NUM] = {0, 0, NUM_MAX};

        for (int i = 0; i < BLOCK_SIZE; i++) {
            aux[INDEX_SUM] += data[start + i];
            
            aux[INDEX_MAX] = max(aux[INDEX_MAX], data[start + i]);

            aux[INDEX_MIN] = min(aux[INDEX_MIN], data[start + i]);
        }

        atomicAdd(&results[INDEX_SUM], aux[INDEX_SUM]);
        atomicMax(&results[INDEX_MAX], aux[INDEX_MAX]);
        atomicMin(&results[INDEX_MIN], aux[INDEX_MIN]);
    }
}

__global__ void kernel_full_atomics(int *data, int *results) {
    int index = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    atomicAdd(&results[INDEX_SUM], data[index]);
    atomicMax(&results[INDEX_MAX], data[index]);
    atomicMin(&results[INDEX_MIN], data[index]);
}

int main(void) {
    int expResults[INDEX_NUM];
    
    int *data = NULL;
    hipMallocManaged(&data, ITEMS_NUM * sizeof(int));
    if (data == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
        return 1;
    }
    
    // generate data and expected result
    expResults[INDEX_SUM] = 0;
    expResults[INDEX_MAX] = 0;
    expResults[INDEX_MIN] = NUM_MAX;
    
    for(int i = 0; i < ITEMS_NUM; i++) {
        // each generated number is lower than NUM_MAX as value
        data[i] = rand() % NUM_MAX;
        
        expResults[INDEX_SUM] += data[i];
        
        expResults[INDEX_MAX] = (data[i] > expResults[INDEX_MAX]) ?
            data[i] : expResults[INDEX_MAX];
        
        expResults[INDEX_MIN] = (data[i] < expResults[INDEX_MIN]) ?
            data[i] : expResults[INDEX_MIN];
    }
    
    int *results = NULL;
    hipMallocManaged(&results, INDEX_NUM * sizeof(int));
    if (results == 0) {
        cout << "[HOST] Couldn't allocate memory\n";
    	return 1;
    }
   
    // compute 10 times the results
    for(int i = 0; i < 10; i++) {
        
        // init
        results[INDEX_SUM] = 0;
        results[INDEX_MAX] = 0;
        results[INDEX_MIN] = NUM_MAX;
        
#ifdef NO_ATOMIC
        kernel_no_atomics<<< 1 , 1 >>> (data, results);
        cudaDeviceSynchronize();
#endif

#ifdef PARTIAL_ATOMIC
        kernel_partial_atomics<<< ITEMS_NUM / 256 , 1 >>> (data, results);
        cudaDeviceSynchronize();
#endif

#ifdef FULL_ATOMIC
        kernel_full_atomics<<< ITEMS_NUM / 256 , 256 >>> (data, results);
        cudaDeviceSynchronize();
#endif
    }
    
    cout << "SUM: " << results[INDEX_SUM] << endl;
    if(results[INDEX_SUM] != expResults[INDEX_SUM]) {
        cout << "Failed, SUM should be " << expResults[INDEX_SUM] << endl;
    }
    
    cout << "MAX: " << results[INDEX_MAX] << endl;
    if(results[INDEX_MAX] != expResults[INDEX_MAX]) {
        cout << "Failed, MAX should be " << expResults[INDEX_MAX] << endl;
    }
    
    cout << "MIN: " << results[INDEX_MIN] << endl;
    if(results[INDEX_MIN] != expResults[INDEX_MIN]) {
        cout << "Failed, MIN should be " << expResults[INDEX_MIN] << endl;
    }
    
    hipFree(results);
    return 0;
}
