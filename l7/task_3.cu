#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

#define BUF_2M		(2 * 1024 * 1024)
#define BUF_32M		(32 * 1024 * 1024)

int main(void) {
    hipSetDevice(0);

    int *host_array_a = 0;
    int *host_array_b = 0;

    int *device_array_a = 0;
    int *device_array_b = 0;
    int *device_array_c = 0;

    host_array_a = (int *)calloc(sizeof(int), BUF_32M);
    host_array_b = (int *)calloc(sizeof(int), BUF_32M);

    hipMalloc(&device_array_a, sizeof(int) * BUF_32M);
    hipMalloc(&device_array_b, sizeof(int) * BUF_32M);
    hipMalloc(&device_array_c, sizeof(int) * BUF_2M);

    // Check for allocation errors
    if (host_array_a == 0 || host_array_b == 0 || 
        device_array_a == 0 || device_array_b == 0 || 
        device_array_c == 0) {
        printf("[*] Error!\n");
        return 1;
    }

    for (int i = 0; i < BUF_32M; ++i) {
        host_array_a[i] = i % 32;
        host_array_b[i] = i % 2;
    }

    printf("Before swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    hipMemcpy(device_array_a, host_array_a, sizeof(int) * BUF_32M, hipMemcpyHostToDevice);
    hipMemcpy(device_array_b, host_array_b, sizeof(int) * BUF_32M, hipMemcpyHostToDevice);

    for (int i = 0; i < BUF_32M; i += BUF_2M) {
        hipMemcpy(device_array_c, device_array_b + i, BUF_2M * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemcpy(device_array_b + i, device_array_a + i, BUF_2M * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemcpy(device_array_a + i, device_array_c, BUF_2M * sizeof(int), hipMemcpyDeviceToDevice);
    }

    hipMemcpy(host_array_a, device_array_a, sizeof(int) * BUF_32M, hipMemcpyDeviceToHost);
    hipMemcpy(host_array_b, device_array_b, sizeof(int) * BUF_32M, hipMemcpyDeviceToHost);

    printf("\nAfter swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    free(host_array_a);
    free(host_array_b);

    hipFree(device_array_a);
    hipFree(device_array_b);
    hipFree(device_array_c);
    return 0;
}