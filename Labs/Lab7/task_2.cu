#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;
    hipError_t err;

    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    host_array_a = (float *)calloc(sizeof(float), N);
    host_array_b = (float *)calloc(sizeof(float), N);
    host_array_c = (float *)calloc(sizeof(float), N);

    DIE(host_array_a == NULL, "malloc(host_array_a)");
    DIE(host_array_b == NULL, "malloc(host_array_a)");
    DIE(host_array_c == NULL, "malloc(host_array_a)");

    err = hipMalloc(&device_array_a, N * sizeof(float));
    DIE(err != hipSuccess || device_array_a == NULL,
		"hipMalloc(device_array_a)");

    err = hipMalloc(&device_array_b, N * sizeof(float));
    DIE(err != hipSuccess || device_array_a == NULL,
		"hipMalloc(device_array_a)");

    err = hipMalloc(&device_array_c, N * sizeof(float));
    DIE(err != hipSuccess || device_array_a == NULL,
		"hipMalloc(device_array_a)");

    fill_array_float(host_array_a, N);
    fill_array_random(host_array_b, N);

    err = hipMemcpy(device_array_a, host_array_a, N * sizeof(float), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "hipMemcpy(host_array_a)");

	err = hipMemcpy(device_array_b, host_array_b, N * sizeof(float), hipMemcpyHostToDevice);
	DIE(err != hipSuccess, "hipMemcpy(host_array_b)");

    const int block_size = 256;
    int num_blocks = N / block_size;

    if (N % block_size) num_blocks++;

    add_arrays<<<num_blocks, block_size>>>(device_array_a, device_array_b, device_array_c, N);
    hipDeviceSynchronize();

    hipMemcpy(host_array_c, device_array_c, N * sizeof(float), hipMemcpyDeviceToHost);

    check_task_2(host_array_a, host_array_b, host_array_c, N);

    free(host_array_a);
    free(host_array_b);
    free(host_array_c);

    hipFree(device_array_a);
    hipFree(device_array_b);
    hipFree(device_array_c);
    return 0;
}